
#include <hip/hip_runtime.h>
extern "C"
__global__ void compress_expand (
  unsigned char* src,
  unsigned char* dst,
  int threshold,
  int src_step,
  int dst_step,
  int width,
  int height
) {
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x > width || y > height)
    return;

  const int src_p = (y * src_step) + (x * 3);
  const int dst_p = (y * dst_step) + (x * 3);
//  /*B*/ dst[dst_p + 0] = src[src_p + 0] * 0.1;
//  /*G*/ dst[dst_p + 1] = src[src_p + 1] * 0.1;
//  /*R*/ dst[dst_p + 2] = src[src_p + 2] * 0.1;

  /*B*/ dst[dst_p + 0] = 255;
  /*G*/ dst[dst_p + 1] = threshold;
  /*R*/ dst[dst_p + 2] = 255;
}