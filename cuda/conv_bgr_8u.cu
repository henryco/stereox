
#include <hip/hip_runtime.h>
extern "C"
__global__ void conv (
  unsigned char* src,
  unsigned char* dst,
  int src_step,
  int dst_step,
  int width,
  int height
) {
  const int col = blockIdx.x * blockDim.x + threadIdx.x;
  const int row = blockIdx.y * blockDim.y + threadIdx.y;

  const int src_p = (row * src_step) + (col * 3);
  const int dst_p = (row * dst_step) + (col * 3);

  /*B*/ dst[dst_p + 0] = src[src_p + 0] * 0.5;
  /*G*/ dst[dst_p + 1] = src[src_p + 1] * 0.5;
  /*R*/ dst[dst_p + 2] = src[src_p + 2] * 0.5;
}