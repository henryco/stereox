
#include <hip/hip_runtime.h>
extern "C"
__global__ void test (
  unsigned char* dst,
  int dst_step,
  int width,
  int height
) {
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x > width || y > height)
    return;

  const int dst_p = (y * dst_step) + (x * 3);
  /*B*/ dst[dst_p + 0] = 255;
  /*G*/ dst[dst_p + 1] = 0;
  /*R*/ dst[dst_p + 2] = 0;

//  const int dst_p = x + y * (width * height);
//  dst[dst_p] = 200;
}