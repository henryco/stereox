
#include <hip/hip_runtime.h>
extern "C"
__global__ void vector_add(int n, int *a, int *b, int *c) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for( int i = index; i < n; i+= stride )
  c[i] = a[i] + b[i];
}